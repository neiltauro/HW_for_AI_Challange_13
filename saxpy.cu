// saxpy.cu (after adding event timing)

#include <hip/hip_runtime.h>
#include <iostream>

__global__
void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}

float saxpyAndTime(int N, float a, float *h_x, float *h_y) {
    float *d_x, *d_y;

    size_t size = N * sizeof(float);
   hipMalloc(&d_x, size);
   hipMalloc(&d_y, size);
   hipMemcpy(d_x, h_x, size,hipMemcpyHostToDevice);
   hipMemcpy(d_y, h_y, size,hipMemcpyHostToDevice);

    // Timing with CUDA events
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    saxpy<<<blocks, threads>>>(N, a, d_x, d_y);
   hipEventRecord(stop);
   hipEventSynchronize(stop);

    float milliseconds = 0;
   hipEventElapsedTime(&milliseconds, start, stop);

    // Cleanup
   hipEventDestroy(start);
   hipEventDestroy(stop);
   hipMemcpy(h_y, d_y, size,hipMemcpyDeviceToHost);
   hipFree(d_y);
   hipFree(d_x);

    return milliseconds;
}

int main()
{
    float a = 2.0f;

    for (int exp = 15; exp <= 25; ++exp) {
        int N = 1 << exp;

        float *h_x = new float[N];
        float *h_y = new float[N];
        for (int i = 0; i < N; ++i) {
            h_x[i] = 1.0f;
            h_y[i] = 2.0f;
        }

        float milliseconds = saxpyAndTime(N, a, h_x, h_y);
        std::cout << N << "," << milliseconds << "ms" <<std::endl;

        delete[] h_y;
        delete[] h_x;
    }

    return 0;
}
